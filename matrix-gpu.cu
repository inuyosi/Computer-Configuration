
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 2048

#define BSX 64
#define BSY 64

void printMatrix(float a[][N],const char *str){
#if (N<=16)
        printf("==== %s ====\n", str);
        for (int i=0; i<N; i++){
                for (int j=0; j<N; j++){
                        printf("%4.0f ", a[i][j]);
                }
                printf("\n");
        }
#endif
}

void initMatrix(float a[][N]){
        for (int i=0; i<N; i++){
                for (int j=0; j<N; j++){
                        a[i][j] = (int)((rand()/(RAND_MAX + 1.0))*10);
                }
        }
}

__global__ void mulMatrix(float (*a)[N],float (*b)[N], float (*c)[N]){
        int xid = blockIdx.x * blockDim.x + threadIdx.x;
        int yid = blockIdx.y * blockDim.y + threadIdx.y;
        if (xid < N && yid < N){
                c[yid][xid] = 0.0;
                for(int k=0; k<N; k++){
                        c[yid][xid] += a[yid][k] * b[k][xid];
                }
        }
}

int main(void){
        float a[N][N],b[N][N],c[N][N],cs[N][N];
        float (*adev)[N],(*bdev)[N],(*cdev)[N];

        initMatrix(a);
        initMatrix(b);
        printMatrix(a, "a");
        printMatrix(b, "b");

//ここからGPU版行列積

hipMalloc(&adev, sizeof(float)*N*N);
hipMalloc(&bdev, sizeof(float)*N*N);
hipMalloc(&cdev, sizeof(float)*N*N);

hipMemcpy(adev, a, sizeof(float)*N*N, hipMemcpyDeviceToHost);
hipMemcpy(bdev, b, sizeof(float)*N*N, hipMemcpyDeviceToHost);

dim3 bdim(BSX,BSY);
dim3 gdim(N/BSX, N/BSY);

printf("\nGPU上の行列積を開始します\n");
mulMatrix<<<gdim, bdim>>>(adev, bdev, cdev);
hipDeviceSynchronize();

hipMemcpy(c, cdev, sizeof(float)*N*N, hipMemcpyDeviceToHost);
printMatrix(c, "c");

hipFree(adev);
hipFree(bdev);
hipFree(cdev);

//ここまでGPU版行列積
printMatrix(cs, "cs");

return 0;
}
