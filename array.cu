
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 8

void printArray(int a[],int size,const char *str){
        printf("%s: ",str);
        for (int i=0;i<size;i++){
                printf("%4d ",a[i]);
        }
        printf("\n");
}

void initArray(int a[], int size){
        for (int i=0;i<size;i++){
                a[i] = (int)((rand()/(RAND_MAX + 1.0)) * 100);
        }
}

__global__ void addArray(int *a, int *b, int *c){
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        if(id < N){
                c[id] = a[id] + b[id];
        }
}

int main(void){
        int a[N],b[N],c[N];
        int *adev,*bdev,*cdev;

hipMalloc(&adev,sizeof(int)*N);
hipMalloc(&bdev,sizeof(int)*N);
hipMalloc(&cdev,sizeof(int)*N);

initArray(a,N);
initArray(b,N);
printArray(a, N, "a");
printArray(b, N, "b");

hipMemcpy(adev, a, sizeof(int)*N, hipMemcpyHostToDevice);
hipMemcpy(bdev, b, sizeof(int)*N, hipMemcpyHostToDevice);

addArray<<<4, 4>>>(adev,bdev,cdev);
hipDeviceSynchronize();

hipMemcpy(cdev, c, sizeof(int)*N, hipMemcpyDeviceToHost);
printArray(c, N, "c");

hipFree(adev);
hipFree(bdev);
hipFree(cdev);

return 0;
}
