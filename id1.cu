
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void f(void){
        int myid = blockIdx.x * blockDim.x + threadIdx.x;

        printf("myid=%-2d,bDim=(%d,%d,%d),bIdx=(%d,%d,%d),tIdx=(%d,%d,%d)\n",
                myid,
                blockDim.x,blockDim.y,blockDim.z,
                blockIdx.x,blockIdx.y,blockIdx.z,
                threadIdx.x,threadIdx.y,threadIdx.z
);
}

int main(void){
        f<<<3, 4>>>();
        hipDeviceSynchronize();
        return 0;
}
